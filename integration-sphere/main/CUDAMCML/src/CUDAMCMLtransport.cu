#include "hip/hip_runtime.h"
/*	This file is part of CUDAMCML.

    CUDAMCML is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMCML is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMCML.  If not, see <http://www.gnu.org/licenses/>.*/

// forward declaration of the device code
template <int ignoreAdetection> __global__ void MCd(MemStruct);
__device__ float rand_MWC_oc(unsigned long long*,unsigned int*);
__device__ float rand_MWC_co(unsigned long long*,unsigned int*);
__device__ void LaunchPhoton(PhotonStruct*,unsigned long long*, unsigned int*);
__global__ void LaunchPhoton_Global(MemStruct);
__device__ void Spin(PhotonStruct*, float,unsigned long long*,unsigned int*);
__device__ unsigned int Reflect(PhotonStruct*, int, unsigned long long*, unsigned int*);
__device__ unsigned int PhotonSurvive(PhotonStruct*, unsigned long long*, unsigned int*);
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add);

template <int ignoreAdetection> __global__ void MCd(MemStruct DeviceMem)
{
    //Block index
    int bx=blockIdx.x;

    //Thread index
    int tx=threadIdx.x;	


    //First element processed by the block
    int begin=NUM_THREADS_PER_BLOCK*bx;
	

    
	unsigned long long int x=DeviceMem.x[begin+tx];//coherent
	unsigned int a=DeviceMem.a[begin+tx];//coherent

	float s;	//step length

	unsigned int index, w, index_old;
	index_old = 0;
	w = 0;
	unsigned int w_temp;
	
	PhotonStruct p = DeviceMem.p[begin+tx];


	int new_layer;
	
	//First, make sure the thread (photon) is active
	unsigned int ii = 0;
	if(!DeviceMem.thread_active[begin+tx]) ii = NUMSTEPS_GPU;

	for(;ii<NUMSTEPS_GPU;ii++) //this is the main while loop
	{
		if(layers_dc[p.layer].mutr!=FLT_MAX)
			s = -__logf(rand_MWC_oc(&x,&a))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		else
			s = 100.0f;//temporary, say the step in glass is 100 cm.
		
		//Check for layer transitions and in case, calculate s
		new_layer = p.layer;
		if(p.z+s*p.dz<layers_dc[p.layer].z_min){new_layer--; s = __fdividef(layers_dc[p.layer].z_min-p.z,p.dz);} //Check for upwards reflection/transmission & calculate new s
		if(p.z+s*p.dz>layers_dc[p.layer].z_max){new_layer++; s = __fdividef(layers_dc[p.layer].z_max-p.z,p.dz);} //Check for downward reflection/transmission

		p.x += p.dx*s;
		p.y += p.dy*s;
		p.z += p.dz*s;

		if(p.z>layers_dc[p.layer].z_max)p.z=layers_dc[p.layer].z_max;//needed?
		if(p.z<layers_dc[p.layer].z_min)p.z=layers_dc[p.layer].z_min;//needed?

		if(new_layer!=p.layer)
		{
			// set the remaining step length to 0
			s = 0.0f;  
 
			if(Reflect(&p,new_layer,&x,&a)==0u)//Check for reflection
			{ // Photon is transmitted
				if(new_layer == 0)
				{ //Diffuse reflectance
					index = __float2int_rz(acosf(-p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr+min(__float2int_rz(__fdividef(sqrtf(p.x*p.x+p.y*p.y),det_dc[0].dr)),(int)det_dc[0].nr-1);
					AtomicAddULL(&DeviceMem.Rd_ra[index], p.weight);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
				if(new_layer > *n_layers_dc)
				{	//Transmitted
					index = __float2int_rz(acosf(p.dz)*2.0f*RPI*det_dc[0].na)*det_dc[0].nr+min(__float2int_rz(__fdividef(sqrtf(p.x*p.x+p.y*p.y),det_dc[0].dr)),(int)det_dc[0].nr-1);
					AtomicAddULL(&DeviceMem.Tt_ra[index], p.weight);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
			}
		}

		//w=0;

		if(s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered)
			w_temp = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			p.weight -= w_temp;
			
			
			//w = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			//p.weight -= w;
			
			if(ignoreAdetection == 0) // Evaluated at compiletime!
			{
				index = (min(__float2int_rz(__fdividef(p.z,det_dc[0].dz)),(int)det_dc[0].nz-1)*det_dc[0].nr+min(__float2int_rz(__fdividef(sqrtf(p.x*p.x+p.y*p.y),det_dc[0].dr)),(int)det_dc[0].nr-1) );
				if(index == index_old)
				{
					w += w_temp;
					//p.weight -= __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight)); 
				}
				else// if(w!=0)
				{
					AtomicAddULL(&DeviceMem.A_rz[index_old], w);
					index_old = index;
					w = w_temp;
				}

			}

			Spin(&p,layers_dc[p.layer].g,&x,&a);
		}




		if(!PhotonSurvive(&p,&x,&a)) // Check if photons survives or not
		{
			if(atomicAdd(DeviceMem.num_terminated_photons,1u) < (*num_photons_dc-NUM_THREADS))
			{	// Ok to launch another photon
				LaunchPhoton(&p,&x,&a);//Launch a new photon
			}
			else
			{	// No more photons should be launched. 
				DeviceMem.thread_active[begin+tx] = 0u; // Set thread to inactive
				ii = NUMSTEPS_GPU;				// Exit main loop
			}
			
		}
	}//end main for loop!
	if(ignoreAdetection == 1 && w!=0)
		AtomicAddULL(&DeviceMem.A_rz[index_old], w);
	
	__syncthreads();//necessary?

	//save the state of the MC simulation in global memory before exiting
	DeviceMem.p[begin+tx] = p;	//This one is incoherent!!!
	DeviceMem.x[begin+tx] = x; //this one also seems to be coherent
	

}//end MCd




__device__ void LaunchPhoton(PhotonStruct* p, unsigned long long* x, unsigned int* a)
{
	// We are currently not using the RNG but might do later
	//float input_fibre_radius = 0.03;//[cm]
	//p->x=input_fibre_radius*sqrtf(rand_MWC_co(x,a));

	p->x  = 0.0f;
	p->y  = 0.0f;
	p->z  = 0.0f;
	p->dx = 0.0f;
	p->dy = 0.0f;
	p->dz = 1.0f;

	p->layer = 1;
	p->weight = *start_weight_dc; //specular reflection!

}

__global__ void LaunchPhoton_Global(MemStruct DeviceMem)//PhotonStruct* pd, unsigned long long* x, unsigned int* a)
{
	int bx=blockIdx.x;
    int tx=threadIdx.x;	

    //First element processed by the block
    int begin=NUM_THREADS_PER_BLOCK*bx;

	PhotonStruct p;
	unsigned long long int x=DeviceMem.x[begin+tx];//coherent
	unsigned int a=DeviceMem.a[begin+tx];//coherent

	LaunchPhoton(&p,&x,&a);

	//__syncthreads();//necessary?
	DeviceMem.p[begin+tx]=p;//incoherent!?
}


__device__ void Spin(PhotonStruct* p, float g, unsigned long long* x, unsigned int* a)
{
	float cost, sint;	// cosine and sine of the 
						// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
						// azimuthal angle psi. 
	float temp;

	float tempdir=p->dx;

	//This is more efficient for g!=0 but of course less efficient for g==0
	temp = __fdividef((1.0f-(g)*(g)),(1.0f-(g)+2.0f*(g)*rand_MWC_co(x,a)));//Should be close close????!!!!!
	cost = __fdividef((1.0f+(g)*(g) - temp*temp),(2.0f*(g)));
	if(g==0.0f)
		cost = 2.0f*rand_MWC_co(x,a) -1.0f;//Should be close close??!!!!!

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rand_MWC_co(x,a),&sinp,&cosp);// spin psi [0-2*PI)
	
	temp = sqrtf(1.0f - p->dz*p->dz);

	if(temp==0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost,p->dz*cost);
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp),temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp),temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp=rsqrtf(p->dx*p->dx+p->dy*p->dy+p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;
}// end Spin

			

__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, unsigned long long* x, unsigned int* a)
{
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)


	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);

	if(n1==n2)//refraction index matching automatic transmission and no direction change
	{	
		p->layer = new_layer;
		return 0u;
	}

	if(n1>n2 && n2*n2<n1*n1*(1-cos_angle_i*cos_angle_i))//total internal reflection, no layer change but z-direction mirroring
	{
		p->dz *= -1.0f;
		return 1u; 
	}

	if(cos_angle_i==1.0f)//normal incident
	{		
		r = __fdividef((n1-n2),(n1+n2));
		if(rand_MWC_co(x,a)<=r*r)
		{
			//reflection, no layer change but z-direction mirroring
			p->dz *= -1.0f;
			return 1u;
		}
		else
		{	//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
		}
	}
	
	//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
	// save a few multiplications, calculate cos_angle_i^2;
	float e = __fdividef(n1*n1,n2*n2)*(1.0f-cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
	r=2*sqrtf((1.0f-cos_angle_i*cos_angle_i)*(1.0f-e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
	e=e+(cos_angle_i*cos_angle_i)*(1.0f-2.0f*e);//Update the value of e
	r = e*__fdividef((1.0f-e-r),((1.0f-e+r)*(e+r)));//Calculate r	

	if(rand_MWC_co(x,a)<=r)
	{ 
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		return 1u;
	}
	else
	{	
		// Transmission, update layer and direction
		r = __fdividef(n1,n2);
		e = r*r*(1.0f-cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1-e) ,p->dz);
		p->layer = new_layer;
		return 0u;
	}

}

__device__ unsigned int PhotonSurvive(PhotonStruct* p, unsigned long long* x, unsigned int* a)
{	//Calculate wether the photon survives (returns 1) or dies (returns 0)

	if(p->weight>WEIGHTI) return 1u; // No roulette needed
	if(p->weight==0u) return 0u;	// Photon has exited slab, i.e. kill the photon

	if(rand_MWC_co(x,a)<CHANCE)
	{
		p->weight = __float2uint_rn(__fdividef((float)p->weight,CHANCE));
		return 1u;
	}

	//else
	return 0u;
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add)
{
	if(atomicAdd((unsigned int*)address,add)+add<add)
		atomicAdd(((unsigned int*)address)+1,1u);
}
